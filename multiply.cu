#include "hip/hip_runtime.h"
/*
  multiply.cu -- Matrix multiplication testbench - by Cody Rivera
*/

#include <cstdio>
#include <cmath>
#include <cstdlib>
#include "hip/hip_runtime.h"
#include "hipblas.h"

#include "multiply.cuh"

#include "kernels_mtsm2_2.cuh"


#define EPS 10e-3
#define MAX_TILES 255

/*
  Helper functions
 */


#include "parameters.cuh"



// Based on https://randomascii.wordpress.com/2012/02/25/comparing-floating-point-numbers-2012-edition/
bool approxEqual(double A, double B,
                 double maxRelDiff = EPS)
{
    // Calculate the difference.
    double diff = fabs(A - B);
    A = fabs(A);
    B = fabs(B);
    // Find the largest
    double largest = (B > A) ? B : A;

    if (diff <= largest * maxRelDiff)
        return true;
    return false;
}


template<typename FloatType>
bool matrixCompare(const FloatType* A, const FloatType* B,
                   unsigned int m, unsigned int n,
                   unsigned int& iFail, unsigned int& jFail)
{
    FloatType aVal, bVal;
    bool b = true;
    // Cache-friendly comparison pattern
    for (unsigned int j = 0; j < n && b; j++)
    {
        for (unsigned int i = 0; i < m && b; i++)
        {
            aVal = A[i + (j * m)];
            bVal = B[i + (j * m)];
            if (!approxEqual(aVal, bVal, EPS))
            {
                iFail = i;
                jFail = j;
                b = false;
            }
        }
    }
    return b;
}

template<typename FloatType>
void reportTestSuccess(const char* testName, double GFLOPs, double totalGFLOPs)
{
    printf("%s succeeded: %g GFLOPs, %g GFLOPs acc. for transfers\n", testName, GFLOPs, totalGFLOPs);
}

template<typename FloatType>
void reportTestFailure(const char* testName,
                       const FloatType* orig, const FloatType* cand,
                       unsigned int leadDim,
                       unsigned int iFail, unsigned int jFail)
{
    double oVal = (double)orig[iFail + (jFail * leadDim)];
    double cVal = (double)cand[iFail + (jFail * leadDim)];
    fprintf(stderr, "%s failed: Original[%u, %u] = %.6f != Candidate[%u, %u] = %.6f\n",
            testName, iFail, jFail, oVal, iFail, jFail, cVal);
}

template<typename FloatType>
double getGFLOPs(double time, unsigned int m, unsigned int n, unsigned int k)
{
    double instCount = ((double) m * (double) n * (double) k) / 10e9;
    double timeSeconds = time / 1000;
    return instCount / timeSeconds;
}



/*
  floatTSM2 and doubleTSM2 -- Wrappers around the kernels that select
  the optimal kernel.

  Currently only optimal for Nvidia V100

  Parameter Choice for V100:
  
  t1 := 128
  
  Single Precision: n ~= t2, t3 := 32
  Double Precision: n ~= t2, t3 := 16 if m < 10240, and t3 := 12 otherwise
  
 */

void floatTSM2(const float* devA, const float* devB, float* devC,
               const unsigned int m, const unsigned int n,
               const unsigned int k)
{
    int blocks = (m / FLOAT_T1) + 1;
    blocks = (blocks > 65536) ? 65536 : blocks;

    if (n <= 2)
    {
        floatTSM2Kernel<FLOAT_T1, 2, 32><<<blocks, FLOAT_T1>>>(devA, devB, devC, m, n, k);
    }
    else if (n <= 4)
    {
        floatTSM2Kernel<FLOAT_T1, 4, 32><<<blocks, FLOAT_T1>>>(devA, devB, devC, m, n, k);
    }
    else if (n <= 6)
    {
        floatTSM2Kernel<FLOAT_T1, 6, 32><<<blocks, FLOAT_T1>>>(devA, devB, devC, m, n, k);
    }
    else if (n <= 8)
    {
        floatTSM2Kernel<FLOAT_T1, 8, 32><<<blocks, FLOAT_T1>>>(devA, devB, devC, m, n, k);
    }
    else
    {
        floatTSM2Kernel<FLOAT_T1, 16, 32><<<blocks, FLOAT_T1>>>(devA, devB, devC, m, n, k);
    }
    // Since CUBLAS starts beating TSM2 at 16, there is no need to include another kernel
}

void doubleTSM2(const double* devA, const double* devB, double* devC,
               const unsigned int m, const unsigned int n,
                const unsigned int k)
{
    int blocks = (m / DOUBLE_T1) + 1;
    blocks = (blocks > 65536) ? 65536 : blocks;
    
    if (n <= 2)
    {
        if (m < 20480)
        {
            doubleTSM2Kernel<DOUBLE_T1, 2, 16><<<blocks, DOUBLE_T1>>>(devA, devB, devC, m, n, k);
        }
        else
        {
            doubleTSM2Kernel<DOUBLE_T1, 2, 12><<<blocks, DOUBLE_T1>>>(devA, devB, devC, m, n, k);
        }
    }
    else if (n <= 4)
    {
        if (m < 20480)
        {
            doubleTSM2Kernel<DOUBLE_T1, 4, 16><<<blocks, DOUBLE_T1>>>(devA, devB, devC, m, n, k);
        }
        else
        {
            doubleTSM2Kernel<DOUBLE_T1, 4, 12><<<blocks, DOUBLE_T1>>>(devA, devB, devC, m, n, k);
        }
    }
    else if (n <= 6)
    {
        if (m < 20480)
        {
            doubleTSM2Kernel<DOUBLE_T1, 6, 16><<<blocks, DOUBLE_T1>>>(devA, devB, devC, m, n, k);
        }
        else
        {
            doubleTSM2Kernel<DOUBLE_T1, 6, 12><<<blocks, DOUBLE_T1>>>(devA, devB, devC, m, n, k);
        }
    }
    else if (n <= 8)
    {
        if (m < 20480)
        {
            doubleTSM2Kernel<DOUBLE_T1, 8, 16><<<blocks, DOUBLE_T1>>>(devA, devB, devC, m, n, k);
        }
        else
        {
            doubleTSM2Kernel<DOUBLE_T1, 8, 12><<<blocks, DOUBLE_T1>>>(devA, devB, devC, m, n, k);
        }
    }
    else if (n <= 16)
    {
        if (m < 20480)
        {
            doubleTSM2Kernel<DOUBLE_T1, 16, 16><<<blocks, DOUBLE_T1>>>(devA, devB, devC, m, n, k);
        }
        else
        {
            doubleTSM2Kernel<DOUBLE_T1, 16, 12><<<blocks, DOUBLE_T1>>>(devA, devB, devC, m, n, k);
        }
    }
    else
    {
        doubleTSM2Kernel<DOUBLE_T1, 32, 12><<<blocks, DOUBLE_T1>>>(devA, devB, devC, m, n, k);
    }    
}







/*
  Executes the kernels
 */
template<>
bool runKernels(const float* A, const float* B, float* C,
                const unsigned int m, const unsigned int n,
                const unsigned int k)
{
    // Candidate for C -- Used by GPU kernels
    float* candC;
    // Device memory
    float* devA, * devB, * devC;
    // Events used for timing
    hipEvent_t start, end, startTotal, endTotal;
    float time, timeTotal;

    printf("Multiplying matrix A[%u, %u] by matrix B[%u, %u]\n\n", m, k, k, n); 

    // Allocates new memory
    candC = (float*)malloc(m * n * sizeof(float));
    if (candC == NULL)
    {
        fprintf(stderr, "Not enough memory\n");
        return false;
    }
    
    cudaErrchk(hipMalloc((float**)&devA, m * k * sizeof(float)));
    cudaErrchk(hipMalloc((float**)&devB, k * n * sizeof(float)));
    cudaErrchk(hipMalloc((float**)&devC, m * n * sizeof(float)));
    
    
    // Inits CUDA events
    cudaErrchk(hipEventCreate(&start));
    cudaErrchk(hipEventCreate(&end));
    cudaErrchk(hipEventCreate(&startTotal));
    cudaErrchk(hipEventCreate(&endTotal));
    
    // Runs CUBLAS call
    hipblasHandle_t handle;
    cublasErrchk(hipblasCreate(&handle));
    
    float one = 1;
    float zero = 0;

    cudaErrchk(hipEventRecord(startTotal));
    
    // Cuda Memory Copy
    cudaErrchk(hipMemcpy(devA, A, m * k * sizeof(float), hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy(devB, B, k * n * sizeof(float), hipMemcpyHostToDevice));

    cudaErrchk(hipEventRecord(start));
    cublasErrchk(hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                              m, n, k,
                              &one, devA, HIP_R_32F, m, devB, HIP_R_32F, k,
                              &zero, devC, HIP_R_32F, m, HIP_R_32F, HIPBLAS_GEMM_DEFAULT));
    cudaErrchk(hipEventRecord(end));
    
    // Copies result back
    cudaErrchk(hipMemcpy(C, devC, m * n * sizeof(float), hipMemcpyDeviceToHost));

    cudaErrchk(hipEventRecord(endTotal));
    cudaErrchk(hipDeviceSynchronize());
    cudaErrchk(hipEventElapsedTime(&time, start, end));
    cudaErrchk(hipEventElapsedTime(&timeTotal, startTotal, endTotal));
    printf("cublas time consume:%f,\t total time: %f\n", time, timeTotal);
    reportTestSuccess<float>("CUBLAS Test", getGFLOPs<float>(time, m, n, k), getGFLOPs<float>(timeTotal, m, n, k)); 

    cublasErrchk(hipblasDestroy(handle));
    

    // Runs kernels
    // Failure flag
    bool status;
    // Failure indices
    unsigned int iFail, jFail;

    // Clear result matrix
    cudaErrchk(hipMemset(devC, 0, m * n * sizeof(float)));
    cudaErrchk(hipEventRecord(startTotal));
    
    // Cuda Memory Copy
    cudaErrchk(hipMemcpy(devA, A, m * k * sizeof(float), hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy(devB, B, k * n * sizeof(float), hipMemcpyHostToDevice));

    int blocks = (k / FLOAT_T1) + 1;
    blocks = (blocks > 65536) ? 65536 : blocks;

    cudaErrchk(hipEventRecord(start));
    #ifdef SINGLE_PARAM
    floatTSM2Kernel<FLOAT_T1, FLOAT_T2, FLOAT_T3><<<blocks, FLOAT_T1>>>(devA, devB, devC, m, n, k);
    #else
    floatTSM2(devA, devB, devC, m, n, k);
    #endif
    cudaErrchk(hipGetLastError());
    cudaErrchk(hipEventRecord(end));
    
    // Copies result back
    cudaErrchk(hipMemcpy(candC, devC, m * n * sizeof(float), hipMemcpyDeviceToHost));

    cudaErrchk(hipEventRecord(endTotal));
    cudaErrchk(hipDeviceSynchronize());
    cudaErrchk(hipEventElapsedTime(&time, start, end));
    cudaErrchk(hipEventElapsedTime(&timeTotal, startTotal, endTotal));
    printf("tsm time consume: %f,\t total time: %f\n", time, timeTotal);
    status = matrixCompare<float>(C, candC, m, n, iFail, jFail);
    if (status)
    {
        reportTestSuccess<float>("TSM2 Kernel Test", 
                                 getGFLOPs<float>(time, m, n, k), 
                                 getGFLOPs<float>(timeTotal, m, n, k)); 
    }
    else
    {
        reportTestFailure<float>("TSM2 Kernel Test", C, candC, m, iFail, jFail);
    }
        
       

    
    cudaErrchk(hipEventDestroy(start));
    cudaErrchk(hipEventDestroy(end));
    cudaErrchk(hipEventDestroy(startTotal));
    cudaErrchk(hipEventDestroy(endTotal));
    free(candC);
    cudaErrchk(hipFree(devA));
    cudaErrchk(hipFree(devB));
    cudaErrchk(hipFree(devC));
    
    return true;
}




template<>
bool runKernels(const double* A, const double* B, double* C,
                const unsigned int m, const unsigned int n,
                const unsigned int k)
{
    // Candidate for C -- Used by GPU kernels
    double* candC;
    // Device memory
    double* devA, * devB, * devC;
    // Events used for timing
    hipEvent_t start, end, startTotal, endTotal;
    float time, timeTotal;

    printf("Multiplying matrix A[%u, %u] by matrix B[%u, %u]\n\n", m, k, k, n); 

    // Allocates new memory
    candC = (double*)malloc(m * n * sizeof(double));
    if (candC == NULL)
    {
        fprintf(stderr, "Not enough memory\n");
        return false;
    }
    
    cudaErrchk(hipMalloc((double**)&devA, m * k * sizeof(double)));
    cudaErrchk(hipMalloc((double**)&devB, k * n * sizeof(double)));
    cudaErrchk(hipMalloc((double**)&devC, m * n * sizeof(double)));
    
    
    // Inits CUDA events
    cudaErrchk(hipEventCreate(&start));
    cudaErrchk(hipEventCreate(&end));
    cudaErrchk(hipEventCreate(&startTotal));
    cudaErrchk(hipEventCreate(&endTotal));
    
    // Runs CUBLAS call
    hipblasHandle_t handle;
    cublasErrchk(hipblasCreate(&handle));
    
    double one = 1;
    double zero = 0;

    cudaErrchk(hipEventRecord(startTotal));
    
    // Cuda Memory Copy
    cudaErrchk(hipMemcpy(devA, A, m * k * sizeof(double), hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy(devB, B, k * n * sizeof(double), hipMemcpyHostToDevice));

    cudaErrchk(hipEventRecord(start));
    cublasErrchk(hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                              m, n, k,
                              &one, devA, HIP_R_64F, m, devB, HIP_R_64F, k,
                              &zero, devC, HIP_R_64F, m, HIP_R_64F, HIPBLAS_GEMM_DEFAULT));
    cudaErrchk(hipEventRecord(end));
    
    // Copies result back
    cudaErrchk(hipMemcpy(C, devC, m * n * sizeof(double), hipMemcpyDeviceToHost));

    cudaErrchk(hipEventRecord(endTotal));
    cudaErrchk(hipDeviceSynchronize());
    cudaErrchk(hipEventElapsedTime(&time, start, end));
    cudaErrchk(hipEventElapsedTime(&timeTotal, startTotal, endTotal));
    reportTestSuccess<double>("CUBLAS Test", getGFLOPs<double>(time, m, n, k), getGFLOPs<double>(timeTotal, m, n, k)); 

    cublasErrchk(hipblasDestroy(handle));
    

    
    
    // Runs kernel
    // Failure flag
    bool status;
    // Failure indices
    unsigned int iFail, jFail;


    // If a TSM

    // Clear result matrix
    cudaErrchk(hipMemset(devC, 0, m * n * sizeof(double)));
    cudaErrchk(hipEventRecord(startTotal));
    
    // Cuda Memory Copy
    cudaErrchk(hipMemcpy(devA, A, m * k * sizeof(double), hipMemcpyHostToDevice));
    cudaErrchk(hipMemcpy(devB, B, k * n * sizeof(double), hipMemcpyHostToDevice));

    int blocks = (k / DOUBLE_T1) + 1;
    blocks = (blocks > 65536) ? 65536 : blocks;

    cudaErrchk(hipEventRecord(start));
    #ifdef SINGLE_PARAM
    doubleTSM2Kernel<DOUBLE_T1, DOUBLE_T2, DOUBLE_T3><<<blocks, DOUBLE_T1>>>(devA, devB, devC, m, n, k);
    #else
    doubleTSM2(devA, devB, devC, m, n, k);
    #endif
    cudaErrchk(hipGetLastError());
    cudaErrchk(hipEventRecord(end));
    
    // Copies result back
    cudaErrchk(hipMemcpy(candC, devC, m * n * sizeof(double), hipMemcpyDeviceToHost));

    cudaErrchk(hipEventRecord(endTotal));
    cudaErrchk(hipDeviceSynchronize());
    cudaErrchk(hipEventElapsedTime(&time, start, end));
    cudaErrchk(hipEventElapsedTime(&timeTotal, startTotal, endTotal));
    status = matrixCompare<double>(C, candC, m, n, iFail, jFail);
    if (status)
    {
        reportTestSuccess<double>("TSM2 Kernel Test", 
                                  getGFLOPs<double>(time, m, n, k), 
                                  getGFLOPs<double>(timeTotal, m, n, k)); 
    }
    else
    {
        reportTestFailure<double>("TSM2 Kernel Test", C, candC, m, iFail, jFail);
    }



    cudaErrchk(hipEventDestroy(start));
    cudaErrchk(hipEventDestroy(end));
    cudaErrchk(hipEventDestroy(startTotal));
    cudaErrchk(hipEventDestroy(endTotal));
    free(candC);
    cudaErrchk(hipFree(devA));
    cudaErrchk(hipFree(devB));
    cudaErrchk(hipFree(devC));
    
    return true;
}